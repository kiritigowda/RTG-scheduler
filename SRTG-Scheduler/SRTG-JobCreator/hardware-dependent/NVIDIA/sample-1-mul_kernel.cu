#include "hip/hip_runtime.h"
/*
 *      Author: Kiriti Nagesh Gowda
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

/*
hipError_t hipFuncSetCacheConfig(reinterpret_cast<const void*>(const char * func), enum hipFuncCache_t
cacheConfig)
*/

// Functions
void Check_CUDA_Error(const char *);
void mul(float *, float *, float *, float, float, float);
void randomInit(float *, int);

// Kernel
__global__ void matrixmul(float *Md, float *Nd, float *Pd, float width,
                          float width_blk, float height_blk, float width_M,
                          float width_N, float height_M, int m, int n) {
  // Block Index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread Index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int Row = by * width_blk + ty;
  int Col = bx * height_blk + tx;

  float pValue = 0;

  if (Col < (int)width_N && Row < (int)height_M) {
    for (int i = 0; i < width; i++) {
      float Melement = Md[Row * (int)width_M + i];
      float Nelement = Nd[i * (int)width_N + Col];

      pValue += Melement * Nelement;
    }
    Pd[Row * (int)width_N + Col] = pValue;
  }
}

int main(void) {
  float *M, *N, *P, *K; // Host Matrices
  float *Md, *Nd, *Pd;  // Device Matrices

  float height_M, width_M; // height and width of matrix M
  float height_N, width_N; // height and width of matrix N

  srand(2006); // setting limit for rand function

  // printf("enter the height and width of matrix M\n");
  // scanf("%f %f",&height_M,&width_M);
  height_M = 32;
  width_M = 32;

  // printf("enter the heigth and width of matrix N\n");
  // scanf("%f %f", &height_N, &width_N);\

  height_N = 32;
  width_N = 32;

  float size_M, size_N, size_P;

  size_M = height_M * width_M;
  size_N = height_N * width_N;
  size_P = height_M * width_N;

  // check whether the given matrix sizes are compatable fr multiplication
  float width = width_M;

  if (width_M == height_N) {

    // Allocate space for the matrices in host memory
    M = (float *)malloc(size_M * sizeof(float));
    N = (float *)malloc(size_N * sizeof(float));
    P = (float *)malloc(size_P * sizeof(float));
    K = (float *)malloc(size_P * sizeof(float));

    // Initialize the data to the matrices  (some random data)
    randomInit(M, size_M);
    randomInit(N, size_N);

    // Allocate memory for the matrices on the GPU
    hipMalloc((void **)&Md, size_M * sizeof(float));
    hipMalloc((void **)&Nd, size_N * sizeof(float));
    hipMalloc((void **)&Pd, size_P * sizeof(float));

    // Transfer Data From host to Device
    hipMemcpy(Md, M, size_M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, size_N * sizeof(float), hipMemcpyHostToDevice);

    // block dimensions
    float height_blk, width_blk;

    // printf("enter the Dimensions of the block\n");
    // scanf("%f %f",&width_blk,&height_blk);

    height_blk = 16;
    width_blk = 16;

    dim3 threads(width_blk, height_blk);

    float x = ceil(width_N / threads.x);
    float y = ceil(height_M / threads.y);

    printf("x=%f\n y=%f\n", x, y);

    dim3 grid(1, 1);

    // Timer
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // kernel Invocation
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    // hipDeviceSetCacheConfig( hipFuncCachePreferShared );
    // hipDeviceSetCacheConfig( hipFuncCachePreferNone );

    matrixmul<<<grid, threads>>>(Md, Nd, Pd, width, width_blk, height_blk,
                                 width_M, width_N, height_M);
    Check_CUDA_Error("Kernel Execution Failed!");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Processing time GPU: %f(ms) \n", time);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Transferring the result data from DEVICE to HOST
    hipMemcpy(P, Pd, size_P * sizeof(float), hipMemcpyDeviceToHost);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // multiplication on host
    mul(M, N, K, height_M, width_N, width_M);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Processing time CPU: %f(ms) \n", time);

    /*
    for(int i=0; i<size_P;i++)
    {
      if(P[i]!= K[i])
      printf("Error at P[%d] = %.5f ------------- K[%d]=%.5f\n",i,P[i],i,K[i]);
    }
    */

    // Free Device Memory
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);

    // Free Host Memory
    free(M);
    free(N);
    free(P);
    free(K);
  } else {
    printf("the sizes of the matrix you entered are not compatable for "
           "multiplication\n");
  }
}

// Fucntions
void Check_CUDA_Error(const char *message) {
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    // fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

// Matrix multiplication on HOST
void mul(float *M, float *N, float *K, float height_M, float width_N,
         float width_M) {
  for (int i = 0; i < height_M; i++) {
    for (int j = 0; j < width_N; j++) {
      float sum = 0;

      for (int k = 0; k < width_M; k++) {
        float a = M[i * (int)width_M + k];
        float b = N[k * (int)width_N + j];
        sum += a * b;
      }
      K[i * (int)width_N + j] = sum;
    }
  }
}

// Initializing the matrices using rand function
void randomInit(float *data, int size) {
  for (int i = 0; i < size; ++i)
    data[i] = rand() / (float)RAND_MAX;
}
